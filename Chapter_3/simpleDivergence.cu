#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

__global__ void mathKernel1(float *c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float a, b;
  a = b = 0.0f;

  if (tid % 2 == 0) {
    a = 100.0f;
  } else {
    b = 200.0f;
  }

  c[tid] = a + b;
}

__global__ void mathKernel2(float *c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float a, b;
  a = b = 0.0f;

  if ((tid / warpSize) % 2 == 0) {
    a = 100.0f;
  } else {
    b = 200.0f;
  }

  c[tid] = a + b;
}

__global__ void mathKernel3(float *c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  bool ipred = (tid % 2 == 0);
  if (ipred) {
    ib = 200.0f;
  }
  if (!ipred) {
    ib = 200.0f;
  }
  c[tid] = ia + ib;
}

__global__ void warmingup(float *c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float a, b;
  a = b = 0.0f;

  if ((tid / warpSize) % 2 == 0) {
    a = 100.0f;
  } else {
    b = 200.0f;
  }

  c[tid] = a + b;
}

double seconds()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (double)tp.tv_sec + (double)tp.tv_usec*1e-6;
}

int main(int argc, char *argv[])
{
  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

  // set up data size
  int size = 64;
  int blocksize = 64;
  if (argc > 1) blocksize = atoi(argv[1]);
  if (argc > 2) size = atoi(argv[2]);
  printf("Data size %d\n", size);

  // set up execution configuration
  dim3 block(blocksize, 1);
  dim3 grid((size+block.x-1)/block.x, 1);
  printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

  // allocate gpu memory
  float *d_C;
  size_t nBytes = size*sizeof(float);
  hipMalloc(&d_C, nBytes);

  // run a warmup kernel to remove overhead
  double iStart, iElaps;
  hipDeviceSynchronize();
  iStart = seconds();
  warmingup<<< grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  printf("warmup         <<< %4d %4d >>> elapsed %lf sec\n", grid.x, block.x, iElaps);

  // run kernel 1
  iStart = seconds();
  mathKernel1<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  printf("mathKernel1    <<< %4d %4d >>> elapsed %lf sec\n", grid.x, block.x, iElaps);

  // run kernel 2
  iStart = seconds();
  mathKernel2<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  printf("mathKernel2    <<< %4d %4d >>> elapsed %lf sec\n", grid.x, block.x, iElaps);

  // run kernel 3
  iStart = seconds();
  mathKernel3<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  printf("mathKernel3    <<< %4d %4d >>> elapsed %lf sec\n", grid.x, block.x, iElaps);

  hipFree(d_C);
  hipDeviceReset();

  exit(EXIT_SUCCESS);
}