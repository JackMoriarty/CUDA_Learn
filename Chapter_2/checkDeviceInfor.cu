#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char *argv[])
{
  printf("%s Starting...\n", argv[0]);

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
	   (int)error_id, hipGetErrorString(error_id));
    printf("Result = FALL\n");
    exit(EXIT_FAILURE);
  }

  if (deviceCount == 0) {
    printf("There are no available device(s) that support CUDA\n");
  } else {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev = 0, driverVersion = 0, runtimeVersion = 0;
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Device %d: \"%s\"\n", dev, deviceProp.name);

  hipRuntimeGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("\u251c\u2500\u2500 CUDA Driver Version / Runtime Version:          %d.%d / %d.%d\n",
	 driverVersion/1000, (driverVersion%100)/10,
	 runtimeVersion/1000, (runtimeVersion%100)/10);
  printf("\u251c\u2500\u2500 CUDA Capability Major/Minor version number:     %d.%d\n",
	 deviceProp.major, deviceProp.minor);
  printf("\u251c\u2500\u2500 Total amount of global memory:                  %.2f MBytes (%llu bytes)\n",
	 (float)deviceProp.totalGlobalMem/(pow(1024, 0.3)),
	 (unsigned long)deviceProp.totalGlobalMem);
  printf("\u251c\u2500\u2500 GPU Clock rate:                                 %.0f MHz (%0.2f GHz)\n",
	 deviceProp.clockRate*1e-3f, deviceProp.clockRate*1e-6f);
  printf("\u251c\u2500\u2500 Memory Clock rate:                              %.0f MHz\n",
	 deviceProp.memoryClockRate*1e-3f);
  printf("\u251c\u2500\u2500 Memory Bus Width:                               %d-bit\n",
	 deviceProp.memoryBusWidth);
  if (deviceProp.l2CacheSize) {
    printf("\u251c\u2500\u2500 L2 Cache Size:                                  %d bytes\n",
	   deviceProp.l2CacheSize);
  }
  printf("\u251c\u2500\u2500 Max Texture Dimension Size (x,y,z)              "
	 "1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
	 deviceProp.maxTexture1D,
	 deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
	 deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1],
	 deviceProp.maxTexture3D[2]);
  printf("\u251c\u2500\u2500 Max Layered Texture Size (dim) x layers          "
	 "1D=(%d) x %d, 2D=(%d,%d) x %d\n",
	 deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
	 deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
	 deviceProp.maxTexture2DLayered[2]);
  printf("\u251c\u2500\u2500 Total amount of const memory:                   %lu bytes\n", deviceProp.totalConstMem);
  printf("\u251c\u2500\u2500 Total amount of shared memory per block:        %lu bytes\n",
	 deviceProp.sharedMemPerBlock);
  printf("\u251c\u2500\u2500 Total number of registers available per block:  %d\n",
	 deviceProp.regsPerBlock);
  printf("\u251c\u2500\u2500 Warp size:                                      %d\n", deviceProp.warpSize);
  printf("\u251c\u2500\u2500 Maximum number of threads per multiprocessor:   %d\n",
	 deviceProp.maxThreadsPerMultiProcessor);
  printf("\u251c\u2500\u2500 Maximum number of threads per block:            %d\n",
	 deviceProp.maxThreadsPerBlock);
  printf("\u251c\u2500\u2500 Maximum sizes of each dimension of a block:     %d x %d x %d\n",
	 deviceProp.maxThreadsDim[0],
	 deviceProp.maxThreadsDim[1],
	 deviceProp.maxThreadsDim[2]);
  printf("\u251c\u2500\u2500 Maximum sizes of each dimension of a grid:      %d x %d x %d\n",
	 deviceProp.maxGridSize[0],
	 deviceProp.maxGridSize[1],
	 deviceProp.maxGridSize[2]);
  printf("\u2514\u2500\u2500 Maximum memory pitch:                           %lu bytes\n", deviceProp.memPitch);
  
  exit(EXIT_SUCCESS);
}