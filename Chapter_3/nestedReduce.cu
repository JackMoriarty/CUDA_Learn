#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n)
{
  // set thread ID
  unsigned int tid = threadIdx.x;
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x;

  // boundary check
  if (idx >= n) return;

  // in-place reduction in global memory
  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    if ((tid % (2*stride)) == 0) {
      idata[tid] += idata[tid + stride];
    }
    
    // synchronize within block
    __syncthreads();
  }

  // write result for this block to global memory
  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

int recursiveReduce(int *data, const int size)
{
  // terminate check
  if (size == 1) return data[0];

  // renew the stride
  int stride = size / 2;

  // in-place reduction
  for (int i = 0; i < stride; i++) {
    data[i] += data[i+stride];
  }

  // call recursively
  return recursiveReduce(data, stride);
}

__global__ void gpuRecursiveReduce(int *g_idata, int *g_odata, unsigned int isize)
{
  // set thread ID
  unsigned int tid = threadIdx.x;

  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x;
  int *odata = &g_odata[blockIdx.x];

  // stop condition
  if (isize == 2 && tid == 0) {
    g_odata[blockIdx.x] = idata[0] + idata[1];
    return;
  }

  // nested invocation
  int istride = isize >> 1;
  if (istride > 1 && tid < istride) {
    // inplace reduction
    idata[tid] += idata[tid + istride];
  }

  // sync at block level
  __syncthreads();

  // nested invocation
  if (tid == 0) {
    gpuRecursiveReduce<<<1, istride>>>(idata, odata, istride);

    // sync all child grids launched in this block
    hipDeviceSynchronize();
  }

  // sync at block level again
  __syncthreads();
}

__global__ void gpuRecursiveReduceNosync(int *g_idata, int *g_odata, unsigned int isize)
{
  // set thread ID
  unsigned int tid = threadIdx.x;

  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x;
  int *odata = &g_odata[blockIdx.x];

  // stop condition
  if (isize == 2 && tid == 0) {
    g_odata[blockIdx.x] = idata[0] + idata[1];
    return ;
  }

  // nested invocation
  int istride = isize >> 1;
  if (istride > 1 && tid < istride) {
    // inplace reduction
    idata[tid] += idata[tid + istride];
  }

  // nested invocation
  if (tid == 0) {
    gpuRecursiveReduceNosync<<<1, istride>>>(idata, odata, istride);
  }
}

__global__ void gpuRecursiveReduce2(int *g_idata, int *g_odata, unsigned int iStride, const int iDim)
{
  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * iDim;

  // stop condition
  if (iStride == 1 && threadIdx.x == 0) {
    g_odata[blockIdx.x] = idata[0] + idata[1];
    return ;
  }

  // nested invocation
  idata[threadIdx.x] += idata[threadIdx.x + iStride];

  // nested invocation
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    gpuRecursiveReduce2<<<1, iStride/2>>>(g_idata, g_odata, iStride/2, iDim);
  }
}

double seconds()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (double)tp.tv_sec*1e3 + (double)tp.tv_usec*1.e-3;
}

int main(int argc, char *argv[])
{
  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("%s starting reduction at ", argv[0]);
  printf("device %d: %s ", dev, deviceProp.name);
  hipSetDevice(dev);

  bool bResult = false;

  // initialization
  int size = 1 << 24;  // total number of elements to reduce
  printf("    with array size %d ", size);

  // execution configuration
  int blocksize = 512;  // initial block size
  if (argc > 1) {
    blocksize = atoi(argv[1]);  // block size form command line argument
  }
  dim3 block(blocksize, 1);
  dim3 grid((size+block.x-1)/block.x, 1);
  printf("grid %d block %d\n", grid.x, block.x);

  // allocate host memory
  size_t bytes = size * sizeof(int);
  int *h_idata = (int *)malloc(bytes);
  int *h_odata = (int *)malloc(grid.x*sizeof(int));
  int *tmp = (int *)malloc(bytes);

  // initialize the array
  for (int i = 0; i < size; i++) {
    // mask off high 2 bytes to force max number to 255
    h_idata[i] = (int)(rand() & 0xFF);
  }
  memcpy(tmp, h_idata, bytes);

  double iStart, iElaps;
  int gpu_sum = 0;

  // allocate device memory
  int *d_idata = NULL;
  int *d_odata = NULL;
  hipMalloc((void **)&d_idata, bytes);
  hipMalloc((void **)&d_odata, grid.x*sizeof(int));

  //cpu reduction
  iStart = seconds();
  int cpu_sum = recursiveReduce(tmp, size);
  iElaps = seconds() - iStart;
  printf("cpu reduce                          elapsed %8d ms cpu_sum: %d\n", iElaps, cpu_sum);

  // kernel warmup: reduceNeighbored
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = seconds();
  reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu Warmup                          elapsed %8.4lf ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

  // kernel 1: reduceNeighbored
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = seconds();
  reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu Neighbored                      elapsed %8.4lf ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

  // kernel 2: GPURecursiveReduce
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = seconds();
  gpuRecursiveReduce<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu gpuRecursiveReduce              elapsed %8.4lf ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

  // kernel 3: GPURecursiveReduceNosync
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = seconds();
  gpuRecursiveReduceNosync<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu GPURecursiveReduceNosync        elapsed %8.4lf ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

  // kernel 4: GPURecursiveReduce2
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = seconds();
  gpuRecursiveReduce2<<<grid, block>>>(d_idata, d_odata, block.x/2, block.x);
  hipDeviceSynchronize();
  iElaps = seconds() - iStart;
  hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu GPURecursiveReduce2             elapsed %8.4lf ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);
 

  // free host memory
  free(h_idata);
  free(h_odata);
  free(tmp);

  // free device memory
  hipFree(d_idata);
  hipFree(d_odata);

  // check the  results
  bResult = (gpu_sum == cpu_sum);
  if (!bResult)
    printf("Test failed!\n");
  
  return 0;
}